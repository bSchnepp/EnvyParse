#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>

#include <wchar.h>

#include <hip/hip_runtime.h>
#include <ctype.h>

typedef struct Ngram
{
	uint16_t Length;
	const wchar_t *Content;
}Ngram;

typedef struct Word
{
	uint16_t Length;
	Ngram *Content;
}Word;

/* isspace doesn't compile for device code. */
#define IS_WHITESPACE(x) (x == ' ' || x == '\n' || x == '\r' || x == '\t' || \
			x == '\v' || x == '\f')

/*
 * Copies the entirety of a file to GPU memory.
 *
 * @return void
 *
 * @param Name The name of the file to open.
 * @param Dst A pointer to an unallocated void pointer which
 *	  will be updated with the result of hipMalloc and
 * 	  the entire content of the file.
 */
void ReadFile(const char *Name, void **Dst, uint64_t *OutSize)
{
	FILE *File = fopen(Name, "r");
	if (File == NULL)
	{
		fprintf(stderr, "Could not open file %s\n", Name);
		return;
	}

	fseek(File, 0, SEEK_END);
	uint64_t Length = ftell(File);
	*OutSize = Length;
	rewind(File);

	
	void *Buffer = malloc(Length);
	uint64_t ReadCount = fread(Buffer, 1, Length, File);
	hipMalloc(Dst, Length);
	hipMemcpy(*Dst, Buffer, Length, hipMemcpyHostToDevice);
	free(Buffer);
	fclose(File);
}

__global__ void TestFile(void *Src, void *Spaces, uint64_t Length)
{
	uint64_t Index = blockIdx.x * blockDim.y + threadIdx.x;
	char *SrcC = (char*)(Src);

	if (Index < Length)
	{
		if (IS_WHITESPACE(SrcC[Index]))
		{
			((char*)(Spaces))[Index] = 1;
		} else {
			((char*)(Spaces))[Index] = 0;
		}
	}
}


int main(int argc, char **argv)
{
	void *TxtPtr;
	void *SpaceBfr;
	void *TxtPtrHost;

	if (argc < 2)
	{
		fprintf(stderr, "Error: missing file name to read.\n");
		return -1;
	}
	const char *FName = argv[1]; 

	uint64_t Length;
	ReadFile(FName, &TxtPtr, &Length);
	hipDeviceSynchronize();

	printf("Got length of file %lu\n", Length);
	hipMallocManaged(&SpaceBfr, Length);
	TxtPtrHost = malloc(Length);

	/* This performs the best on my GP102. (11GB VRam) */
	hipMemPrefetchAsync(SpaceBfr, Length, 0);
	TestFile<<<Length, 1>>>(TxtPtr, SpaceBfr, Length);
	hipDeviceSynchronize();
	hipMemcpy(TxtPtrHost, TxtPtr, Length, hipMemcpyDeviceToHost);
	for (uint64_t Index = 0; Index < Length; ++Index)
	{
		if (((char*)SpaceBfr)[Index] == 0)
		{
			printf("%c", ((char*)(TxtPtrHost))[Index]);	
		}
	}
	printf("\n");

	hipFree(TxtPtr);
	hipHostFree(SpaceBfr);
	free(TxtPtrHost);
	return 0;
}
